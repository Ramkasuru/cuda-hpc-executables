#include <iostream>
#include <hip/hip_runtime.h>
#define N 1024

__global__ void reduce_atomic( const float* input, float* result) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        atomicAdd(result, input[i]);    
    }
}
   void check_cuda_error(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    } 
}

int main(){
    size_t size = N * sizeof(float);
    float * h_input = new float[N];

    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f; // Initialize input array with 1.0

    }
    // Allocating memory on the device
    float *d_input, *d_result;
    check_cuda_error(hipMalloc(&d_input, size));
    check_cuda_error(hipMalloc(&d_result, sizeof(float)));

    // Initialize result to zero on the device
    float h_result = 0.0f;

    // Copy data to device
    check_cuda_error(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));
    check_cuda_error(hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice));

    // Kernel launch
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    reduce_atomic<<<numBlocks, blockSize>>>(d_input, d_result);
    check_cuda_error(hipGetLastError());
    check_cuda_error(hipDeviceSynchronize());

    // Copy result back
    check_cuda_error(hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Result from atomic reduction: " << h_result << std::endl;

    // Cleanup
    delete[] h_input;
    hipFree(d_input);
    hipFree(d_result);

    return 0;
}

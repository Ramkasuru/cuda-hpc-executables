#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>  // for atoi

__global__ void reduce_atomic(const float* input, float* result, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        atomicAdd(result, input[i]);
    }
}

void check_cuda_error(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <array_size> <block_size>" << std::endl;
        return 1;
    }

    int N = std::atoi(argv[1]);
    int blockSize = std::atoi(argv[2]);

    if (N <= 0 || blockSize <= 0) {
        std::cerr << "Error: array_size and block_size must be positive integers." << std::endl;
        return 1;
    }

    size_t size = N * sizeof(float);
    float* h_input = new float[N];

    // Initialize input array with 1.0f for testing
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;
    }

    float* d_input;
    float* d_result;

    check_cuda_error(hipMalloc(&d_input, size));
    check_cuda_error(hipMalloc(&d_result, sizeof(float)));

    // Initialize result to zero on device
    float h_result = 0.0f;
    check_cuda_error(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));
    check_cuda_error(hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice));

    // Calculate number of blocks needed
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Launch kernel
    reduce_atomic<<<numBlocks, blockSize>>>(d_input, d_result, N);
    check_cuda_error(hipGetLastError());
    check_cuda_error(hipDeviceSynchronize());

    // Copy result back to host
    check_cuda_error(hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Result from atomic reduction: " << h_result << std::endl;

    // Cleanup
    delete[] h_input;
    hipFree(d_input);
    hipFree(d_result);

    return 0;
}

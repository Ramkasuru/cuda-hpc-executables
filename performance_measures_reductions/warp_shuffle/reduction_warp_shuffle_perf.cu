#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__inline__ __device__ int warpReduceSum(int val) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__global__ void reduce_warp_shuffle(int *input, int *output, int N) {
    int sum = 0;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) sum = input[tid];
    sum = warpReduceSum(sum);
    if ((threadIdx.x % 32) == 0) atomicAdd(output, sum);
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s <num_elements> <threads_per_block>\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    int *h_input = (int*)malloc(N * sizeof(int));
    int h_output = 0;
    for (int i = 0; i < N; ++i) h_input[i] = 1;

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output, &h_output, sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce_warp_shuffle<<<blocks, threadsPerBlock>>>(d_input, d_output, N);

    hipEventRecord(stop);
    hipDeviceSynchronize();

    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum: %d\n", h_output);
    printf("Kernel time (ms): %f\n", ms);

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}


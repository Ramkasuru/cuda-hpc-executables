#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void reduce_atomic(int *input, int *output, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        atomicAdd(output, input[tid]);
    }
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s <num_elements> <threads_per_block>\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    int *h_input = (int*)malloc(N * sizeof(int));
    int h_output = 0;

    // Initialize input with 1s for easy checking
    for (int i = 0; i < N; ++i)
        h_input[i] = 1;

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output, &h_output, sizeof(int), hipMemcpyHostToDevice);

    // Timing setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    reduce_atomic<<<blocks, threadsPerBlock>>>(d_input, d_output, N);

    hipEventRecord(stop);
    hipDeviceSynchronize();

    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum: %d\n", h_output);
    printf("Kernel time (ms): %f\n", ms);

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

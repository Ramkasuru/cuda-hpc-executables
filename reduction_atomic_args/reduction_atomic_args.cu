#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void reduce_atomic(int *input, int *output, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        atomicAdd(output, input[tid]);
    }
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s <num_elements> <threads_per_block>\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    int *h_input = (int*)malloc(N * sizeof(int));
    int h_output = 0;

    // Initialize input with 1s for easy checking
    for (int i = 0; i < N; ++i)
        h_input[i] = 1;

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output, &h_output, sizeof(int), hipMemcpyHostToDevice);

    reduce_atomic<<<blocks, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum: %d\n", h_output);

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    return 0;
}
